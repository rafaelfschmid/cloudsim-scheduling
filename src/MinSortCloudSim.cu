#include <jni.h>
//#include <iostream>
#include "MinSortCloudSim.h"
#include <hip/hip_runtime.h>
#include <moderngpu/kernel_segsort.hxx>

JNIEXPORT void JNICALL Java_MinSortCloudSim_segmented_1sort (JNIEnv * env, jobject obj,
		jfloatArray ma, jintArray ta, jintArray seg, jint t, jint m) {

    float *machines = env->GetFloatArrayElements(ma, 0);
    int *task_index  = env->GetIntArrayElements(ta, 0);
    int *segments = env->GetIntArrayElements(seg, 0);

	int *d_task_index;
	int *d_segments;
	float *d_machines;

	int mem_size_seg = sizeof(int) * (m);
	int mem_size_machines = sizeof(float) * (m * t);
	int mem_size_task_index = sizeof(uint) * (m * t);

	hipMalloc((void **) &d_segments, mem_size_seg);
	hipMalloc((void **) &d_machines, mem_size_machines);
	hipMalloc((void **) &d_task_index, mem_size_task_index);

	//hipEventRecord(start);
	// copy host memory to device
	hipMemcpy(d_segments, segments, mem_size_seg, hipMemcpyHostToDevice);
	hipMemcpy(d_machines, machines, mem_size_machines, hipMemcpyHostToDevice);
	hipMemcpy(d_task_index, task_index, mem_size_task_index, hipMemcpyHostToDevice);

	mgpu::standard_context_t context;
	mgpu::segmented_sort(d_machines, d_task_index, m * t, d_segments, m, mgpu::less_t<float>(), context);

	hipMemcpy(task_index, d_task_index, mem_size_task_index, hipMemcpyDeviceToHost);
	hipMemcpy(machines, d_machines, mem_size_machines, hipMemcpyDeviceToHost);

	env->ReleaseFloatArrayElements(ma, machines, NULL);
	env->ReleaseIntArrayElements(ta, task_index, NULL);


}

JNIEXPORT void JNICALL Java_MinSortCloudSim_segmented_1sort_1desc (JNIEnv * env, jobject obj,
		jfloatArray ma, jintArray ta, jintArray seg, jint t, jint m) {

    float *machines = env->GetFloatArrayElements(ma, 0);
    int *task_index  = env->GetIntArrayElements(ta, 0);
    int *segments = env->GetIntArrayElements(seg, 0);

	int *d_task_index;
	int *d_segments;
	float *d_machines;

	int mem_size_seg = sizeof(int) * (m);
	int mem_size_machines = sizeof(float) * (m * t);
	int mem_size_task_index = sizeof(uint) * (m * t);

	hipMalloc((void **) &d_segments, mem_size_seg);
	hipMalloc((void **) &d_machines, mem_size_machines);
	hipMalloc((void **) &d_task_index, mem_size_task_index);

	//hipEventRecord(start);
	// copy host memory to device
	hipMemcpy(d_segments, segments, mem_size_seg, hipMemcpyHostToDevice);
	hipMemcpy(d_machines, machines, mem_size_machines, hipMemcpyHostToDevice);
	hipMemcpy(d_task_index, task_index, mem_size_task_index, hipMemcpyHostToDevice);

	mgpu::standard_context_t context;
	mgpu::segmented_sort(d_machines, d_task_index, m * t, d_segments, m, mgpu::greater_t<float>(), context);

	hipMemcpy(task_index, d_task_index, mem_size_task_index, hipMemcpyDeviceToHost);
	hipMemcpy(machines, d_machines, mem_size_machines, hipMemcpyDeviceToHost);

	env->ReleaseFloatArrayElements(ma, machines, NULL);
	env->ReleaseIntArrayElements(ta, task_index, NULL);


}
